#include "hip/hip_runtime.h"
// only using global memory
__global__ void slow_add_kernel(float* a, float* b, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_out = new float[N];
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blockSize = 256, gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    slow_add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0; hipEventElapsedTime(&ms, start, stop);
    std::cout << "Slow add took " << ms << " ms\n";

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    delete[] h_a;
    delete[] h_b;
    delete[] h_out;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}