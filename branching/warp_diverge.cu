#include <iostream>
#include <hip/hip_runtime.h>

__global__ void warp_diverge(float* a, float* b, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (i % 2 == 0)
            out[i] = a[i] * 2;
        else
            out[i] = b[i] * 3;
    }
}

int main() {
    int N = 1024 * 1024;
    size_t size = N * sizeof(float);
    float* h_a = new float[N];
    float* h_b = new float[N];
    float* h_out = new float[N];

    for (int i = 0; i < N; ++i) {
        h_a[i] = i * 0.001f;
        h_b[i] = i * 0.002f;
    }

    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    warp_diverge<<<(N + 255)/256, 256>>>(d_a, d_b, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    for (int i = N / 2 - 5; i < N / 2 + 5; ++i)
        std::cout << "out[" << i << "] = " << h_out[i] << std::endl;

    std::cout << "\n Warp-divergent time: " << ms << " ms\n";

    delete[] h_a; delete[] h_b; delete[] h_out;
    hipFree(d_a); hipFree(d_b); hipFree(d_out);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}