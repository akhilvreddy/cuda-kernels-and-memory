#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void apply_tanh(float* x, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = tanhf(x[i]);
    }
}

int main() {
    int N = 1024 * 1024;
    size_t size = N * sizeof(float);
    float* h_x = new float[N];
    float* h_out = new float[N];

    for (int i = 0; i < N; ++i)
        h_x[i] = (i - N / 2) * 0.001f;

    float *d_x, *d_out;
    hipMalloc(&d_x, size);
    hipMalloc(&d_out, size);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    apply_tanh<<<gridSize, blockSize>>>(d_x, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    for (int i = N / 2 - 5; i < N / 2 + 5; ++i)
        std::cout << "tanh(" << h_x[i] << ") = " << h_out[i] << std::endl;

    std::cout << "\n Tanh GPU time: " << ms << " ms\n";

    delete[] h_x; delete[] h_out;
    hipFree(d_x); hipFree(d_out);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}