#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void tanh_abs(float* x, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float val = tanhf(x[i]);
        out[i] = fabsf(val);
    }
}

int main() {
    int N = 1024 * 1024;
    size_t size = N * sizeof(float);
    float* h_x = new float[N];
    float* h_out = new float[N];

    for (int i = 0; i < N; ++i)
        h_x[i] = (i - N / 2) * 0.005f;

    float *d_x, *d_out;
    hipMalloc(&d_x, size);
    hipMalloc(&d_out, size);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    tanh_abs<<<gridSize, blockSize>>>(d_x, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    for (int i = N / 2 - 5; i < N / 2 + 5; ++i)
        std::cout << "abs(tanh(" << h_x[i] << ")) = " << h_out[i] << std::endl;

    std::cout << "\n Fused Tanh+Abs GPU time: " << ms << " ms\n";

    delete[] h_x; delete[] h_out;
    hipFree(d_x); hipFree(d_out);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}