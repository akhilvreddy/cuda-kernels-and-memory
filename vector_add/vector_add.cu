#include <iostream>
#include <hip/hip_runtime.h>

// main CUDA kernel
__global__ void vector_add(float* a, float* b, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = a[i] + b[i];
    }
}

// this is going to be run in colab
int main() {
    int N = 1024;
    size_t size = N * sizeof(float);

    // allocate memory on host (CPU)
    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_out = new float[N];

    // input arrays
    for (int i = 0; i < N; ++i) {
        h_a[i] = float(i);
        h_b[i] = float(N - i);
    }

    // allocate memory on device (GPU)
    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // start kernel with recorded time
    hipEventRecord(start);
    vector_add<<<gridSize, blockSize>>>(d_a, d_b, d_out, N);
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU kernel execution time: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // first 10 results
    for (int i = 0; i < 10; ++i) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_out[i] << std::endl;
    }

    // free memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_out;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return 0;
}