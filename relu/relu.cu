#include <iostream>
#include <hip/hip_runtime.h>

__global__ void relu(float* x, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = x[i] > 0 ? x[i] : 0.0f;
    }
}

int main() {
    int N = 1024;
    size_t size = N * sizeof(float);

    float *h_x = new float[N];
    float *h_out = new float[N];

    // input needs to have both negative and positive values (to show effects of ReLU)
    for (int i = 0; i < N; ++i) {
        h_x[i] = i - 512;  // range: -512 to +511
    }

    float *d_x, *d_out;
    hipMalloc(&d_x, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    // timer (just like vector_add)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    relu<<<gridSize, blockSize>>>(d_x, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // first 10 values
    for (int i = 510; i < 520; ++i) {
        std::cout << "ReLU(" << h_x[i] << ") = " << h_out[i] << std::endl;
    }

    std::cout << "GPU kernel execution time: " << ms << " ms" << std::endl;

    delete[] h_x;
    delete[] h_out;
    hipFree(d_x);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}