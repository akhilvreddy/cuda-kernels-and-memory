// ReLU + Square (just like I talked about in the blog post)

#include <iostream>
#include <hip/hip_runtime.h>

// fused kernel
__global__ void relu_square(float* x, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float val = x[i] > 0 ? x[i] : 0.0f;
        out[i] = val * val;
    }
}

int main() {
    int N = 1024 * 1024;
    size_t size = N * sizeof(float);

    float* h_x = new float[N];
    float* h_out = new float[N];

    for (int i = 0; i < N; ++i)
        h_x[i] = i - N / 2;

    float *d_x, *d_out;
    hipMalloc(&d_x, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEventRecord(start);
    relu_square<<<gridSize, blockSize>>>(d_x, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    for (int i = N / 2 - 5; i < N / 2 + 5; ++i) {
        std::cout << "ReLU^2(" << h_x[i] << ") = " << h_out[i] << std::endl;
    }

    std::cout << "\n Fused GPU ReLU+Square time: " << ms << " ms\n";

    delete[] h_x;
    delete[] h_out;
    hipFree(d_x);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}